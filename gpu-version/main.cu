#include "hip/hip_runtime.h"
#include "camera.cuh"
#include "object.cuh"
#include "material.cuh"
#include "rtweekend.cuh"
#include "texture.cuh"

#include "color.cuh"
#include "parser.hpp"

#include <iostream>
#include <cfloat>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// 计算光线 r 在 world 中的反射结果，最大深度为 depth
__device__ color ray_color(const ray &r, const color &background,
                           hittable **world, int depth, hiprandState *rng) {
    //    UPDATE: 改为递归运算，因为非递归需要用栈来合并最终结果
    ray now = r;
    color accumulated_attenuation(1.0f, 1.0f, 1.0f);
    color accumulated_color(0.0f, 0.0f, 0.0f);
    // UPDATE 将 hittable 与 material 类整合到一起，方便数据传输
    // UPDATE 将递归调用改为循环判断，适应 cuda 的计算
    // UPDATE 仍然将 hittable 与 material 类分开，但是 hittable
    // 在拷贝到显存后需要重新设置 material 的指针
    while (depth > 0) {
        hit_record rec;

        // hittable_list *list = (hittable_list *)(*world);
        // if (list != nullptr) {
        //     printf("successful cast to hittable_list\n");
        //     printf("list size: %d\n", list->len);
        //     sphere *s = (sphere *)list->objects[0];
        //     printf("sphere center: %f %f %f\n", s->center.x(), s->center.y(),
        //            s->center.z());
        //     lambertian *lam = (lambertian *)s->mat_ptr;
        //     solid_color *sc = (solid_color *)lam->albedo;
        //     printf("lambertian albedo: %f %f %f\n", sc->color_value.x(),
        //            sc->color_value.y(), sc->color_value.z());
        // } else {
        //     printf("failed cast to hittable_list\n");
        // }

        if ((*world)->hit(now, 0.001, FLT_MAX, rec)) {
            ray scattered;
            color attenuation;
            color emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.p);

            if (rec.mat_ptr->scatter(now, rec, attenuation, scattered, rng)) {
                accumulated_color += emitted * accumulated_attenuation;
                accumulated_attenuation = accumulated_attenuation * attenuation;
                depth--;
                now = scattered;
                continue;
            } else {
                // 没有反射，返回自发光
                accumulated_color += accumulated_attenuation * emitted;
                break;
            }
        } else {
            // 没有碰到物体，返回环境颜色
            accumulated_color += accumulated_attenuation * background;
            break;
        }
    }

    // 超过最大深度，光线衰减到 0
    return accumulated_color;
}

__global__ void render(int sample, color background, camera **cam,
                       hittable **world, int max_depth, int image_width,
                       int image_height, color *image, hiprandState *states) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int id = y * image_width + x;

    if (x >= image_width)
        return;
    if (y >= image_height)
        return;

    hiprandState *rng = &states[id];

    color res(0, 0, 0);

    // print detail info of cam
    // printf("%d\n", __LINE__);
    // auto cam_data = **cam;
    // printf("%d\n", __LINE__);
    // printf("cam: %f %f %f\n", cam_data.origin.x(), cam_data.origin.y(),
    //        cam_data.origin.z());

    for (int s = 0; s < sample; ++s) {
        //        printf("sample: %d/%d\n", s, sample);
        auto u = float(x + random_float(rng)) / (image_width - 1);
        auto v = float(y + random_float(rng)) / (image_height - 1);
        ray r = (*cam)->get_ray(u, v, rng);
        res += ray_color(r, background, world, max_depth, rng);
    }
    // UPDATE 将除以采样数的操作移动到了 kernel 函数内
    // UPDATE 还是将操作保留在 write_color 函数里吧
    image[id] = res;
}

// UPDATE 复制 PI 的全局变量和 inf 的全局变量到设备内存
void init_constant() {
    constexpr float tmp_inf = std::numeric_limits<float>::infinity();
    //    constexpr float tmp_inf = 1e9;
    const float tmp_pi = acos(-1);
    when("inf: %f, pi: %f\n", tmp_inf, tmp_pi);

    // UPDATE hipMemcpyToSymbol 中设备端的变量是不需要加 & 的
    // UPDATE 使用 define 定义的常量替代常数常量
    // checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(inf), &tmp_inf, sizeof(float)));
    // checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pi), &tmp_pi, sizeof(float)));
}

__global__ void init_random_library(hiprandState *state) {
    int idx = blockIdx.x;
    // 固定种子，方便比较性能
    // UPDATE 更新随机数生成方法
    hiprand_init(idx, 0, 0, &state[idx]);
}

// UPDATE 并行化世界生成
__global__ void random_scene(hittable **list, hittable **world, camera **cam,
                             int image_width, int image_height,
                             hiprandState *states, int num_of_objects) {

    //     UPDATE 添加小型场景进行测试
    if (true) {
        list[0] = new sphere(vec3(0, 0, -1), 0.5,
                             new lambertian(vec3(0.1, 0.2, 0.5)));
        list[1] = new sphere(vec3(0, -100.5, -1), 100,
                             new lambertian(vec3(0.8, 0.8, 0.0)));
        list[2] = new sphere(vec3(1, 0, -1), 0.5,
                             new metal(vec3(0.8, 0.6, 0.2), 0.0));
        list[3] = new sphere(vec3(-1, 0, -1), 0.5, new dielectric(1.5));
        list[4] = new sphere(vec3(-1, 0, -1), -0.45, new dielectric(1.5));
        *world = new hittable_list(list, 5);

        // Camera
        point3 lookfrom(13, 2, 3);
        point3 lookat(0, 0, 0);
        vec3 vup(0, 1, 0);
        auto dist_to_focus = (lookfrom - lookat).length();
        //        *cam = new camera(lookfrom, lookat, vup, 20,
        //        float(image_width) / float(image_height), 0.1,
        //        dist_to_focus);

        *cam = new camera(vec3(-2, 2, 1), vec3(0, 0, -1), vec3(0, 1, 0), 20.0,
                          float(image_width) / float(image_height), 0,
                          dist_to_focus);

        return;
    }

    int id = blockIdx.x;
    auto *rng = &states[id];

    int a = id / 22 - 11;
    int b = id % 22 - 11;
    float choose_mat = random_float(rng);
    vec3 center(a + 0.9 * random_float(rng), 0.2, b + 0.9 * random_float(rng));

    material *sphere_material;

    // 为了保证数量固定，球只要生成了就会加入到世界
    if (choose_mat < 0.8f) {
        auto albedo = color::random(rng) * color::random(rng);
        sphere_material = new lambertian(albedo);
        list[id] = new sphere(center, 0.2, sphere_material);
    } else if (choose_mat < 0.95f) {
        auto albedo = color::random(0.5, 1, rng);
        auto fuzz = random_float(rng);
        sphere_material = new metal(albedo, fuzz);
        list[id] = new sphere(center, 0.2, sphere_material);
    } else {
        sphere_material = new dielectric(1.5);
        list[id] = new sphere(center, 0.2, sphere_material);
    }

    if (id == 0) {
        //        list[num_of_objects - 4] = new sphere(vec3(0, -1000.0, 0),
        //        1000, new lambertian(vec3(0.5, 0.5, 0.5)));
        list[num_of_objects - 4] =
            new sphere(vec3(0, -1000.0, 0), 1000,
                       new lambertian(new checker_texture(
                           color(0.2, 0.3, 0.1), color(0.9, 0.9, 0.9))));
        list[num_of_objects - 3] =
            new sphere(vec3(0, 2, 0), 1.0, new dielectric(1.5));
        list[num_of_objects - 2] = new sphere(
            vec3(-4, 2, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));

        auto difflight = new diffuse_light(color(4, 4, 4));
        auto rect_light = new xy_rect(3, 5, 1, 3, -2, difflight);
        //        list[num_of_objects - 1] = new sphere(vec3(4, 2, 0), 1.0, new
        //        metal(vec3(0.7, 0.6, 0.5), 0.0));;
        auto cylinder_light = new cylinder(0.5, 0, 2, difflight);
        list[num_of_objects - 1] = cylinder_light;

        *world = new hittable_list(list, num_of_objects);

        // Camera
        point3 lookfrom(2, 2, -13);
        point3 lookat(0, 0, 0);
        vec3 vup(0, 1, 0);
        auto dist_to_focus = (lookfrom - lookat).length();
        auto aperture = 0.1;
        *cam = new camera(lookfrom, lookat, vup, 20,
                          float(image_width) / float(image_height), aperture,
                          dist_to_focus);
    }
}

__global__ void free_scene(hittable **list, hittable **world, camera **cam,
                           int num_of_objects) {
    for (int i = 0; i < num_of_objects; i++) {
        delete list[i];
    }
    delete *world;
    delete *cam;
}

std::tuple<hittable **, camera **> get_coded_scene(int image_width,
                                                   int image_height,
                                                   int num_of_objects,
                                                   hiprandState *states) {
    // UPDATE hittable_list 需要从 vector
    // 迁移到数组，使用指针开辟空间，方便在显卡间传输数据 UPDATE hittable_list
    // 从数组迁移到 thrust_vector，数组不方便处理继承问题 UPDATE hittable_list
    // 还是使用了指针实现，并且在显卡上创建 在 cuda
    // 的函数中创建世界和相机，因为要使用 new 创建，不方便使用 malloc
    // 直接创建然后拷贝
    hittable **dev_lists, **dev_world;
    camera **dev_camera;

    checkCudaErrors(
        hipMalloc((void **)&dev_lists, sizeof(hittable *) * num_of_objects));
    checkCudaErrors(hipMalloc((void **)&dev_world, sizeof(hittable *)));
    checkCudaErrors(hipMalloc((void **)&dev_camera, sizeof(camera *)));
    when("Finish the allocation of objects, world, camera\n");

    random_scene<<<num_of_objects, 1>>>(dev_lists, dev_world, dev_camera,
                                        image_width, image_height, states,
                                        num_of_objects);
    when("Finish the creation of world, objects, camera\n");

    return {dev_world, dev_camera};
}
int oldmain(int argc, char *argv[]) {
    // cpu 计时功能
    auto start = clock();
    when("Start counting time\n");

    // Init image
    constexpr auto aspect_ratio = 16.0 / 9.0;
    constexpr int image_width = 800;
    constexpr int image_height = static_cast<int>(image_width / aspect_ratio);
    int samples_per_pixel = 500;
    const int num_of_objects = 5;

    int max_depth = 50;

    // 根据命令行参数设置图像参数
    // UPDATE 删去调整图像长宽的参数
    // for (int i = 0; i < argc; i++) {
    //     if (strcmp(argv[i], "-d") == 0) {
    //         max_depth = atoi(argv[i + 1]);
    //     } else if (strcmp(argv[i], "-spp") == 0) {
    //         samples_per_pixel = atoi(argv[i + 1]);
    //     }
    // }

    const int wrap = 8;
    dim3 grids(image_width / wrap + 1, image_height / wrap + 1);
    dim3 threads(wrap, wrap);

    // 随机化库的初始化操作
    hiprandStateXORWOW_t *states;
    constexpr int num_of_pixels = image_height * image_width;
    checkCudaErrors(
        hipMalloc(&states, sizeof(hiprandStateXORWOW_t) * num_of_pixels));
    when("Finish the memory allocation of random library\n");

    // 随机数生成器的初始化操作
    // UPDATE 将随机数初始化从 1xnum_of_pixels 改为
    // num_of_pixelsx1，前者会超过线程数限制
    init_constant();
    init_random_library<<<num_of_pixels, 1>>>(states);

    // 完成随机数库和常数的初始化
    checkCudaErrors(hipDeviceSynchronize());
    when("Finish the initialization of random library and constants\n");

    // UPDATE 将世界和相机的创建放到函数中
    hittable **dev_world;
    camera **dev_camera;
    std::tie(dev_world, dev_camera) =
        get_coded_scene(image_width, image_height, num_of_objects, states);

    // 分配本地和显卡图像的空间
    static color image[num_of_pixels];
    color *dev_image;
    checkCudaErrors(
        hipMalloc((void **)&dev_image, sizeof(color) * num_of_pixels));
    when("Finish the allocation of image\n");

    // 完成世界、相机、图像内存的初始化
    checkCudaErrors(hipDeviceSynchronize());
    when("Start rendering\n");

    render<<<grids, threads>>>(samples_per_pixel, color(0.3, 0.7, 1.0),
                               dev_camera, dev_world, max_depth, image_width,
                               image_height, dev_image, states);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
    when("Finish rendering\n");

    // 输出
    checkCudaErrors(hipMemcpy(image, dev_image, sizeof(color) * num_of_pixels,
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());
    when("Copying image\n");

    // 重定向输出到 main.ppm
    (void)freopen("main.ppm", "w", stdout);
    printf("P3\n%d %d\n255\n", image_width, image_height);

    for (int j = image_height - 1; j >= 0; --j) {
        for (int i = 0; i < image_width; ++i) {
            write_color(stdout, image[j * image_width + i], samples_per_pixel);
        }
    }

    FILE *fp = fopen("gpu-cuda-version-time.log", "a");
    fprintf(
        fp,
        "basic cuda versions, improve world generate, image width: %d,image "
        "height: %d, max depth: %d, samples per pixel: %d, time: %f s\n",
        image_width, image_height, max_depth, samples_per_pixel,
        (clock() - start) / float(CLOCKS_PER_SEC));
    fclose(fp);

    // 清理退出程序
    // UPDATE 让操作系统去 free 把，free 不动了
    // free_scene<<<1, 1>>>(dev_lists, dev_world, dev_camera, num_of_objects);
    // checkCudaErrors(hipFree(dev_lists));
    // checkCudaErrors(hipFree(dev_world));
    // checkCudaErrors(hipFree(dev_camera));
    // checkCudaErrors(hipFree(dev_image));
    // checkCudaErrors(hipFree(states));
    hipDeviceReset();
    return 0;
}

void output_image(color *image, int image_width, int image_height,
                  int samples_per_pixel, std::string filename) {
    // 重定向输出到 main.ppm
    FILE *fp = fopen(filename.c_str(), "w");
    fprintf(fp, "P3\n%d %d\n255\n", image_width, image_height);

    for (int j = image_height - 1; j >= 0; --j) {
        for (int i = 0; i < image_width; ++i) {
            write_color(fp, image[j * image_width + i], samples_per_pixel);
        }
    }

    fclose(fp);
}

__device__ mytexture *move_to_device(mytexture *src) {
    if (src->type == class_type::solid_color) {
        return new solid_color(((solid_color *)src)->color_value);
    }
    if (src->type == class_type::checker) {
        return new checker_texture(
            move_to_device(((checker_texture *)src)->odd),
            move_to_device(((checker_texture *)src)->even));
    }
    printf("error happend in %s:%d\n", __FILE__, __LINE__);
}

__device__ material *move_to_device(material *src) {
    if (src->type == class_type::lambertian) {
        return new lambertian(move_to_device(((lambertian *)src)->albedo));
    }
    if (src->type == class_type::metal) {
        return new metal(((metal *)src)->albedo, ((metal *)src)->fuzz);
    }
    if (src->type == class_type::dielectric) {
        return new dielectric(((dielectric *)src)->ir);
    }
    if (src->type == class_type::diffuse_light) {
        return new diffuse_light(move_to_device(((diffuse_light *)src)->emit));
    }
    printf("error happend in %s:%d\n", __FILE__, __LINE__);
}

__device__ hittable *move_to_device(hittable *src) {
    if (src->type == class_type::xy_rect) {
        auto tmp = (xy_rect *)src;
        return new xy_rect(tmp->x0, tmp->x1, tmp->y0, tmp->y1, tmp->k,
                           move_to_device(tmp->mp));
    }
    if (src->type == class_type::yz_rect) {
        auto tmp = (yz_rect *)src;
        return new yz_rect(tmp->y0, tmp->y1, tmp->z0, tmp->z1, tmp->k,
                           move_to_device(tmp->mp));
    }
    if (src->type == class_type::xz_rect) {
        auto tmp = (xz_rect *)src;
        return new xz_rect(tmp->x0, tmp->x1, tmp->z0, tmp->z1, tmp->k,
                           move_to_device(tmp->mp));
    }
    if (src->type == class_type::hittable_list) {
        hittable_list *dst = new hittable_list();
        dst->len = ((hittable_list *)src)->len;
        dst->objects = new hittable *[dst->len];
        for (int i = 0; i < dst->len; i++) {
            dst->objects[i] =
                move_to_device(((hittable_list *)src)->objects[i]);
        }
        // return new bvh_node(dst->objects, 0, dst->len);
        return dst;
    }
    if (src->type == class_type::sphere) {
        return new sphere(((sphere *)src)->center, ((sphere *)src)->radius,
                          move_to_device(((sphere *)src)->mat_ptr));
    }
    if (src->type == class_type::cylinder) {
        auto dev_obj =
            new cylinder(((cylinder *)src)->radius, ((cylinder *)src)->zmin,
                         ((cylinder *)src)->zmax,
                         move_to_device(((cylinder *)src)->mat_ptr));
        dev_obj->o2w = ((cylinder *)src)->o2w;
        return dev_obj;
    }
    printf("error happend in %s:%d\n", __FILE__, __LINE__);
}

__global__ void move_to_device(hittable **src, hittable **dst) {
    *dst = move_to_device(*src);
}

int jsonmain(int argc, char *argv[]) {
    hipDeviceSetLimit(hipLimitStackSize, 8192 * 4);
    // cpu 计时功能
    auto start = clock();
    when("Start counting time\n");

    // parse scene file name from argc
    std::string scene_file_name = "sample_scene.json";
    for (int i = 0; i < argc; i++) {
        if (strcmp(argv[i], "-f") == 0) {
            scene_file_name = argv[i + 1];
        }
    }

    scene *world;
    json data;
    std::tie(world, data) = parse_scene(scene_file_name);
    when("Finish parsing scene\n");

    const int wrap = 8;
    dim3 grids(world->width / wrap + 1, world->height / wrap + 1);
    dim3 threads(wrap, wrap);

    hittable **dev_world;
    checkCudaErrors(hipMalloc((void **)&dev_world, sizeof(hittable *)));
    move_to_device<<<1, 1>>>(world->world, dev_world);
    when("Finish the allocation of objects, world, camera\n");

    auto dev_camera = world->cam;
    when("Finish the allocation of objects, world, camera\n");

    // 分配本地和显卡图像的空间
    int num_of_pixels = world->height * world->width;
    color *image;
    checkCudaErrors(
        hipMallocManaged((void **)&image, sizeof(color) * num_of_pixels));
    when("Finish the allocation of image\n");

    // 随机化库的初始化操作
    hiprandStateXORWOW_t *states;
    checkCudaErrors(
        hipMalloc(&states, sizeof(hiprandStateXORWOW_t) * num_of_pixels));
    when("Finish the memory allocation of random library\n");

    // 随机数生成器的初始化操作
    // UPDATE 将随机数初始化从 1xnum_of_pixels 改为
    // num_of_pixelsx1，前者会超过线程数限制
    init_constant();
    init_random_library<<<num_of_pixels, 1>>>(states);

    // 完成随机数库和常数的初始化
    checkCudaErrors(hipDeviceSynchronize());
    when("Finish the initialization of random library and constants\n");

    checkCudaErrors(hipDeviceSynchronize());
    when("Start rendering\n");

    render<<<grids, threads>>>(world->samples_per_pixel, world->background,
                               dev_camera, dev_world, world->max_depth,
                               world->width, world->height, image, states);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
    when("Finish rendering\n");

    output_image(image, world->width, world->height, world->samples_per_pixel,
                 "main.ppm");
    write_image(world->width, world->height, image, world->samples_per_pixel,
                data["output_file"].get<std::string>());
    when("Finish writing image\n");

    hipDeviceReset();
    when("Program finish, cost: %f s\n",
         double(clock() - start) / CLOCKS_PER_SEC);
    return 0;
}

int main(int argc, char *argv[]) {
    // oldmain(argc,argv);
    jsonmain(argc, argv);
}